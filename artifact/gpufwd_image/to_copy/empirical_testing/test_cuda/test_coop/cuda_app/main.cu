#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include <atomic>
#include <thread>
#include <vector>
#include <chrono>
#include <pthread.h>
#include <functional>

// From: https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using std_clock = std::chrono::system_clock;
using sec = std::chrono::duration<double>;
using namespace std;

typedef void (*tfunc)(uint* arg0, uint* arg1, uint* arg2);

vector<string> kernel_names;
vector<tfunc*> kernels;

#include "kernels.h"
#include "kernel_init.h"

__global__ void test_launcher_kernel(uint * x, uint * y, uint* count, tfunc f) {
  f(x,y,count);
}


#define ITERS 20

int main(int argc, char **argv) {

  const unsigned int total_blocks = 65532;
  const unsigned int bufferSize = (total_blocks * sizeof(int));

  int t_num = 0;
  int i_num = 0;
  if (argc == 3) {
    t_num = atoi(argv[1]);
    i_num = atoi(argv[2]);
    printf("executing t: %d, i: %d\n",t_num, i_num);
  }

  uint *dBufferX;
  uint *dBufferY;
  uint *dBufferCounter;
  uint *hBufferX;
  uint *hBufferY;
  uint *hBufferCounter;
  
  gpuErrchk(hipMalloc(&dBufferX, bufferSize)); 
  gpuErrchk(hipMalloc(&dBufferY, bufferSize));
  gpuErrchk(hipMalloc(&dBufferCounter, sizeof(uint)));

  hBufferX = (uint*) malloc(bufferSize);
  hBufferY = (uint*) malloc(bufferSize);
  hBufferCounter = (uint *) malloc(sizeof(uint));

  init_kernels();
  int total_killed = 0;
  tfunc host_function_ptr;

  // From https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#grid-synchronization-cg
  int dev = 0;
  int supportsCoopLaunch = 0;
  hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, dev);
  cout << "supports cooperative launch: " << supportsCoopLaunch << endl;
  if (supportsCoopLaunch == 0) {
    cout << "device doesn't support cooperative launch, exiting" << endl;
    return 1;
  }

  // In case we want to do it iteratively
  //for (int t = t_num; t < kernel_names.size(); t++) {
  for (int t = t_num; t < t_num+1; t++) {
    cout << "running test: " << kernel_names[t] << "\n";
    gpuErrchk(hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(*(kernels[t])), sizeof(tfunc)));

    int success = 0;
    int killed = 0;

    for (int i = i_num; i < i_num+1; i++) {
      
      for (int i = 0; i < total_blocks; i++) {
	hBufferX[i] = 0;
	hBufferY[i] = 0;
      }
      hBufferCounter[0] = 0;
      
      gpuErrchk(hipMemcpy(dBufferX, hBufferX, bufferSize, hipMemcpyHostToDevice));
      gpuErrchk(hipMemcpy(dBufferY, hBufferY, bufferSize, hipMemcpyHostToDevice));
      gpuErrchk(hipMemcpy(dBufferCounter, hBufferCounter, sizeof(uint), hipMemcpyHostToDevice));

      // From: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#grid-synchronization-cg
      /// This will launch a grid that can maximally fill the GPU, on the default stream with kernel arguments
      int numBlocksPerSm = 0;
      // Number of threads my_kernel will be launched with
      int numThreads = 1;
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, dev);
      hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, test_launcher_kernel, numThreads, 0);
      cout << "number of blocks per SM " << numBlocksPerSm << endl;
      
      // launch
      void *kernelArgs[] = { &dBufferX, &dBufferY, &dBufferCounter, &host_function_ptr };

      int total_blocks = deviceProp.multiProcessorCount*numBlocksPerSm;
      while (total_blocks % 6 != 0) {
	total_blocks -= 1;
      }
      cout << "launching with blocks: " << total_blocks << endl;
      dim3 dimBlock(1, 1, 1);
      dim3 dimGrid(total_blocks, 1, 1);

      hipLaunchCooperativeKernel((void *) test_launcher_kernel, dimGrid, dimBlock, kernelArgs);
      
      
      gpuErrchk(hipMemcpy(hBufferCounter, dBufferCounter, sizeof(uint), hipMemcpyDeviceToHost));
      assert(*hBufferCounter == total_blocks);
    }
  }
    
  gpuErrchk(hipFree(dBufferX));
  gpuErrchk(hipFree(dBufferY));
  gpuErrchk(hipFree(dBufferCounter));
  free(hBufferX);
  free(hBufferY);
  free(hBufferCounter);
  return 0;    
}
