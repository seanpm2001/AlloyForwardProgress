#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include <atomic>
#include <thread>
#include <vector>
#include <chrono>
#include <pthread.h>
#include <functional>

// From: https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using std_clock = std::chrono::system_clock;
using sec = std::chrono::duration<double>;
using namespace std;

typedef void (*tfunc)(uint* arg0, uint* arg1, uint* arg2);

vector<string> kernel_names;
vector<tfunc*> kernels;

#include "kernels.h"
#include "kernel_init.h"

__global__ void test_launcher_kernel(uint * x, uint * y, uint* count, tfunc f) {
  f(x,y,count);
}

#define ITERS 20

int main(int argc, char **argv) {

  const unsigned int total_blocks = 1;
  const unsigned int bufferSize = (3 * sizeof(int));

  int t_num = 0;
  int i_num = 0;
  if (argc == 3) {
    t_num = atoi(argv[1]);
    i_num = atoi(argv[2]);
    printf("executing t: %d, i: %d\n",t_num, i_num);
  }

  uint *dBufferX;
  uint *dBufferY;
  uint *dBufferCounter;
  uint *hBufferX;
  uint *hBufferY;
  uint *hBufferCounter;
  
  gpuErrchk(hipMalloc(&dBufferX, bufferSize)); 
  gpuErrchk(hipMalloc(&dBufferY, bufferSize));
  gpuErrchk(hipMalloc(&dBufferCounter, sizeof(uint)));

  hBufferX = (uint*) malloc(bufferSize);
  hBufferY = (uint*) malloc(bufferSize);
  hBufferCounter = (uint *) malloc(sizeof(uint));

  init_kernels();
  int total_killed = 0;
  tfunc host_function_ptr;
  assert(t_num <= kernel_names.size());
  assert(t_num <= kernels.size());

  // In case we want to do it iteratively
  //for (int t = t_num; t < kernel_names.size(); t++) {
  for (int t = t_num; t < t_num+1; t++) {
    cout << "running test: " << kernel_names[t] << "\n";
    gpuErrchk(hipMemcpyFromSymbol(&host_function_ptr, HIP_SYMBOL(*(kernels[t])), sizeof(tfunc)));

    int success = 0;
    int killed = 0;

    //for (int i = 0; i < ITERS; i++) {
    for (int i = i_num; i < i_num+1; i++) {
      //printf("test: %d, i: %d\n",t,i);
      //fflush(stdout);
      
      for (int i = 0; i < total_blocks; i++) {
	hBufferX[i] = 0;
	hBufferY[i] = 0;
      }
      hBufferCounter[0] = 0;
      
      gpuErrchk(hipMemcpy(dBufferX, hBufferX, bufferSize, hipMemcpyHostToDevice));
      gpuErrchk(hipMemcpy(dBufferY, hBufferY, bufferSize, hipMemcpyHostToDevice));
      gpuErrchk(hipMemcpy(dBufferCounter, hBufferCounter, sizeof(uint), hipMemcpyHostToDevice));

      test_launcher_kernel<<<1,3>>>(dBufferX, dBufferY, dBufferCounter, host_function_ptr);
      
      gpuErrchk(hipDeviceSynchronize());
      
      gpuErrchk(hipMemcpy(hBufferCounter, dBufferCounter, sizeof(uint), hipMemcpyDeviceToHost));
      assert(*hBufferCounter == 3);
      //printf("found %d\n", *hBufferCounter);
    }
  }
    
  gpuErrchk(hipFree(dBufferX));
  gpuErrchk(hipFree(dBufferY));
  gpuErrchk(hipFree(dBufferCounter));
  free(hBufferX);
  free(hBufferY);
  free(hBufferCounter);
  return 0;    
}
